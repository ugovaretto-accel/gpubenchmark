#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <string>
#include <stdexcept>
#include <sstream>
#include <cassert>

typedef float real_t;


size_t searchPageLockedLimit( size_t m = 0xffff, size_t M = 0x3fffffff ) {
	assert( m <= M );
	const size_t h = ( M + m ) / 2;
	if( h == m || h > size_t(1E9) ) return h;
	void* p = 0;
	if( hipHostAlloc( &p, h, hipHostMallocDefault ) == hipSuccess ) {
		hipHostFree( p );
		return searchPageLockedLimit( h, M );
	}
	return searchPageLockedLimit( m, h );
}


size_t findPow2PageLockedLimit( size_t s = 65536 ) {
	void* p = 0;
	hipError_t e = hipSuccess;
	while( e == hipSuccess && s > 0 ) {
		e = hipHostAlloc( &p, s, hipHostMallocDefault );
		if( e == hipSuccess ) {
			hipHostFree( p );
			return s;
		}
		s >>= 1;
	}
	return s;
}


template < typename T >
T strTo( const char* str ) {
	if( !str ) throw std::runtime_error( "strTo<T> - NULL srting");
	std::istringstream is( str );
	T v = T();
	is >> v;
	return v;
}

const double GB = 1024 * 1024 * 1024;
double GBs( size_t bytes, double tms ) {
	return ( bytes / GB ) / ( tms / 1000 );
}


hipEvent_t start = hipEvent_t();
hipEvent_t stop  = hipEvent_t();


hipError_t memCpy( void* target, const void* src, size_t bytes, hipMemcpyKind flags, const char* msg ) {
	float elapsed = 0.f;
	hipError_t e = hipEventRecord( start ); if( e != hipSuccess ) return e;
 	e = hipMemcpy( target, src, bytes, flags ); if( e != hipSuccess ) return e;
 	e = hipEventRecord( stop ); if( e != hipSuccess ) return e;
 	e = hipEventSynchronize( stop ); if( e != hipSuccess ) return e;
 	e = hipEventElapsedTime( &elapsed, start, stop ); if( e != hipSuccess ) return e;
 	std::cout << bytes << ',' << msg << ','  << GBs( bytes, elapsed ) << ',' << elapsed << std::endl;
 	return e;
 }


size_t nextStep( size_t prev, size_t val, bool mul ) {
	return mul ? val * prev : prev + val;
}


int main(int argc, char** argv ) {

	size_t beginSize = 0;
	size_t endSize   = 0;
	size_t step      = 0;
	bool   mul       = false;
	int    dev_id    = 0;
			
	if( argc < 5 ) {
		std::cout << "usage: " << argv[ 0 ] << " <min num elements>  <max num elements> <step> [device id]\n";
		return 0;
	} else {		
		beginSize = strTo< size_t >( argv[ 1 ] );
		endSize   = strTo< size_t >( argv[ 2 ] );
		step 	  = strTo< size_t >( argv[ 3 ] );
		if( argc > 4 ) dev_id = strTo< int >( argv[ 4 ] ); 
	}

	if( hipSetDevice( dev_id ) != hipSuccess ) {
		std::cerr << "Cannot set device " << dev_id << std::endl;
		return 1;
	}

	hipSetDeviceFlags( hipDeviceMapHost );
	
        #if 0
	//size_t pll = findPow2PageLockedLimit( 1024 * 1024 * 1024 );
	pll = searchPageLockedLimit( pll, 2 * pll );	
	std::cout <<  "Max power of two page-lockable size (<= 1GB): " 
	          <<  double( pll ) / ( 1024 * 1024 ) << "MB" << std::endl;
	#endif	
	
	hipEventCreate( &start );
	hipEventCreate( &stop  );
 	
 	for( size_t s = beginSize; s < endSize; s = nextStep( s, step, mul ) ) {
 		const size_t NUM_ELEMENTS = s;
 		const size_t SIZE = NUM_ELEMENTS * sizeof( real_t );
 		{
 			real_t* dev_in, *dev_out;
 			hipMalloc( &dev_in, SIZE );
 			hipMalloc( &dev_out, SIZE );
 			memCpy( dev_out, dev_in, SIZE, hipMemcpyDeviceToDevice, "Device to Device"); 		
 			hipFree( dev_in );
 			hipFree( dev_out ); 
 		}
		//
		{
			real_t *dev;
			std::vector< real_t > host( NUM_ELEMENTS );
			hipMalloc( &dev, SIZE );
			memCpy( dev, &host[ 0 ], SIZE, hipMemcpyHostToDevice, "Host to Device"  );
			memCpy( &host[ 0 ], dev, SIZE, hipMemcpyDeviceToHost, "Device to host"  );
			hipFree( dev );
		}
		//
		{
			real_t *dev, *host;
			hipMalloc( &dev, SIZE );
			const hipError_t e =  hipHostAlloc( &host, SIZE, hipHostMallocDefault /*same as cudaMallocHost*/ );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc" << std::endl;
				return 1;
			}
			memCpy( dev, host, SIZE, hipMemcpyHostToDevice, "Host to Device - hipHostMallocDefault" );
			memCpy( host, dev, SIZE, hipMemcpyDeviceToHost, "Device to Host - hipHostMallocDefault" );
			hipFree( dev );
			hipHostFree( host );
		}
		//
		{
			real_t *dev, *host;
			hipMalloc( &dev, SIZE );
			const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocPortable );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc" << std::endl;
				return 1;
			}
			memCpy( dev, host, SIZE, hipMemcpyHostToDevice,  "Host to Device - hipHostMallocPortable" );
			memCpy( host, dev, SIZE, hipMemcpyDeviceToHost,   "Device to Host - hipHostMallocPortable" );
			hipFree( dev );
			hipHostFree( host );
		}
		//
		{
			real_t *dev, *host;
			hipMalloc( &dev, SIZE );
			const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocMapped );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc - mapped" << std::endl;
				return 1;
			}
			memCpy( dev, host, SIZE, hipMemcpyHostToDevice, "Host to Device - hipHostMallocMapped" );
			memCpy( host, dev, SIZE, hipMemcpyDeviceToHost, "Device to Host - hipHostMallocMapped" );
			hipFree( dev );
			hipHostFree( host );
		}
		//
		{
			real_t *dev, *host, *mapped_host;
			hipMalloc( &dev, SIZE );
			const hipError_t e  = hipHostAlloc( &host, SIZE, hipHostMallocMapped );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc - mapped" << std::endl;
				return 1;
			}
			hipHostGetDevicePointer( &mapped_host, host, 0 /*has to be zero "for now" */ );
			memCpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice, "Device(on host) to Device - hipHostMallocMapped" );
			memCpy( mapped_host, dev, SIZE, hipMemcpyDeviceToDevice, "Device to Device(on host) - hipHostMallocMapped" );
			hipFree( dev );
			hipHostFree( host );
		}
		//
		{
			real_t *dev, *host;
			hipMalloc( &dev, SIZE );
			const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocWriteCombined );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc - write combining" << std::endl;
				return 1;
			}
			memCpy( dev, host, SIZE, hipMemcpyHostToDevice, "Host to Device - hipHostMallocWriteCombined" );
			memCpy( host, dev, SIZE, hipMemcpyDeviceToHost, "Device to Host - hipHostMallocWriteCombined" );
			hipFree( dev );
			hipHostFree( host );
		}
		//
		{
			real_t *dev, *host, *mapped_host;
			hipMalloc( &dev, SIZE );
			const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocMapped | hipHostMallocWriteCombined );
			if( e != hipSuccess ) {
				std::cerr << "Error - hipHostAlloc - mapped | write combining" << std::endl;
				return 1;
			}
			hipHostGetDevicePointer( &mapped_host, host, 0 /*has to be zero "for now" */ );
			memCpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice, "Device(on host) to Device - hipHostMallocMapped | hipHostMallocWriteCombined" );
			memCpy( mapped_host, dev, SIZE, hipMemcpyDeviceToDevice, "Device to Device(on host) - hipHostMallocMapped | hipHostMallocWriteCombined" );
			hipFree( dev );
			hipHostFree( host );
		}

		#if CUDART_VERSION >= 4000
		//
		{
                        float elapsed = float();
		 	real_t *dev, *mapped_host;
		 	hipMalloc( &dev, SIZE );
		 	std::vector< real_t > host( NUM_ELEMENTS );	

		 	const hipError_t e = hipHostRegister( &host[ 0 ], SIZE, hipHostMallocMapped );
		 	if( e != hipSuccess ) {
		 		std::cerr << "Error - hipHostRegister - mapped" << std::endl;
		 		return 1;
		 	}
		 	hipHostGetDevicePointer( &mapped_host, &host[ 0 ], 0 /*has to be zero "for now" */ );
			
		 	hipEventRecord( start );
		 		hipMemcpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice );
		 	hipEventRecord( stop );
	 	 	hipEventSynchronize( stop );
	 	 		hipEventElapsedTime( &elapsed, start, stop );
		 	std::cout << "\nHost to device - hipHostRegister - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;	

		 	hipEventRecord( start );
		 		hipMemcpy( &host[ 0 ], dev, SIZE, hipMemcpyDeviceToHost );
		 	hipEventRecord( stop );
	 	 		hipEventSynchronize( stop );
	 	 		hipEventElapsedTime( &elapsed, start, stop );
		 	std::cout << "\nDevice to host - hipHostRegister - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;	

		 	hipFree( dev );
		 	hipHostUnregister( &host[ 0 ] );
		 }
		 #endif
	}

	return 0;
}
