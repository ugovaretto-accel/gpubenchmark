#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <string>
#include <stdexcept>
#include <sstream>
#include <cassert>
#include <malloc.h>
#include "Timer.h"

typedef float real_t;



void* aligned_malloc( size_t size, size_t alignment ) {
    assert( sizeof( size_t ) == sizeof( void* ) );
    const size_t OFFSET = sizeof( size_t ) > alignment ? sizeof( size_t ) : alignment;
    char* ptr = ( char* ) malloc( size + OFFSET );
    char* buffer_start = ptr + OFFSET;
    if( ( size_t ) buffer_start  % alignment != 0 ) {
        buffer_start += alignment - ( ( size_t ) buffer_start ) % alignment;
    }
    size_t* start_pointer = ( size_t* ) ( buffer_start - sizeof( size_t ) );
    *start_pointer = ( size_t ) ptr;  
    std::cout << *start_pointer << std::endl;        
    if( ( size_t ) buffer_start % alignment != 0 ) std::cerr << "ERROR" << std::endl;
    return buffer_start;
}

void aligned_free( void* ptr ) {
    std::cout <<  *( ( size_t* ) ( (char*) ptr - sizeof( size_t ) ) ) << std::endl;
    free( ( void* ) *( (size_t* ) ( (char*) ptr - sizeof( size_t ) ) ) ) ; 
}


size_t findPageLockedLimit( size_t prev = 65536 ) {
    float* p = 0;
    const hipError_t e = hipHostAlloc( &p, prev, hipHostMallocDefault );
    if( e != hipSuccess ) return prev / 2;
    hipHostFree( p );
    return findPageLockedLimit( 2 * prev );
}


template < typename T >
T strTo( const char* str ) {
    if( !str ) throw std::runtime_error( "strTo<T> - NULL srting");
    std::istringstream is( str );
    T v = T();
    is >> v;
    return v;
}

const double GB = 1024 * 1024 * 1024;
double GBs( size_t numElements, double tms ) {
    return ( ( numElements * sizeof( real_t ) ) / GB ) / ( tms / 1000 );
}

//a.exe 4194304 0 16 1 16 128 csv
int main(int argc, char** argv ) {

    size_t NUM_ELEMENTS = 64 * 1024  *1024;
    bool CSV = false;
        
    if( argc < 2 || argc > 3 ) {
        std::cout << "usage: " << argv[ 0 ] << " <num elements>  [csv]\n";
        std::cout << "  using default: num elements= " << NUM_ELEMENTS << std::endl;
    } else {
        NUM_ELEMENTS = strTo< size_t >( argv[ 1 ] );
        std::cout << "NUM ELEMENTS: " << NUM_ELEMENTS << ' ' <<
                      double( NUM_ELEMENTS * sizeof( real_t ) ) / ( 1024*1024 ) <<
                      " MB" << std::endl;
        if( argc == 3 ) { CSV = std::string( argv[ 2 ] ) == "csv"; }
    }

    if( hipSetDevice( 0 ) != hipSuccess ) {
        std::cerr << "Cannot set device" << std::endl;
        return 1;
    }
hipDeviceReset();
hipDeviceSynchronize();
    hipSetDeviceFlags( hipDeviceMapHost );
#if 0       
    std::cout <<     "\nMax power of two page lockable size:                      " 
              << findPageLockedLimit() / ( 1024 * 1024 ) << "MB" << std::endl;
#endif  
            
    const size_t SIZE = sizeof( real_t ) * NUM_ELEMENTS;            
            
    hipEvent_t start = hipEvent_t();
    hipEvent_t stop  = hipEvent_t();
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    float elapsed = 0.f;
    
    //
    {
        real_t* dev_in, *dev_out;
        hipMalloc( &dev_in, SIZE );
        hipMalloc( &dev_out, SIZE );
        
        hipEventRecord( start );
            hipMemcpy( dev_out, dev_in, SIZE, hipMemcpyDeviceToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to device - hipMemcpyDeviceToDevice:              "  << 2 * GBs( NUM_ELEMENTS, elapsed ) << std::endl;
        
        hipFree( dev_in );
        hipFree( dev_out ); 
    }
    //
    {
        real_t *dev;
        //std::vector< real_t > host( NUM_ELEMENTS );
        real_t* host = (real_t*) memalign( 128, SIZE );
        if( (size_t ) host % 16 == 0 ) std::cout << "ALIGNED" << std::endl;
        hipMalloc( &dev, SIZE );
        
        hipEventRecord( start );
        hipMemcpy( dev, &host[ 0 ], SIZE, hipMemcpyHostToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - hipMalloc:                              " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;
        
        hipEventRecord( start );
            hipMemcpy( &host[ 0 ], dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );  
        std::cout << "\nDevice to host - hipMalloc:                              " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;

        hipFree( dev );
    }
    //
    {
        real_t *dev, *host;
        hipMalloc( &dev, SIZE );
        const hipError_t e =  hipHostAlloc( &host, SIZE, hipHostMallocDefault /*same as hipHostMalloc*/ );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc" << std::endl;
            return 1;
        }
        hipEventRecord( start );
            hipMemcpy( dev, host, SIZE, hipMemcpyHostToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost:                           " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;
        
        hipEventRecord( start );
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost:                           " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;

        hipFree( dev );
        hipHostFree( host );
    }
    //
    {
        real_t *dev, *host;
        hipMalloc( &dev, SIZE );
        const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocPortable );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc" << std::endl;
            return 1;
        }
        
        hipEventRecord( start );
            hipMemcpy( dev, host, SIZE, hipMemcpyHostToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost - portable:                " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   

        hipEventRecord( start );   
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost - portable:                " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;

        hipFree( dev );
        hipHostFree( host );
    }
    //
    {
        real_t *dev, *host;
        hipMalloc( &dev, SIZE );
        const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocMapped );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc - mapped" << std::endl;
            return 1;
        }
        
        hipEventRecord( start );
            hipMemcpy( dev, host, SIZE, hipMemcpyHostToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost - mapped:                  " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   
        
        hipEventRecord( start );
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost - mapped:                  " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;       
        
        hipFree( dev );
        hipHostFree( host );
    }
    //
    {
        real_t *dev, *host, *mapped_host;
        hipMalloc( &dev, SIZE );
        const hipError_t e  = hipHostAlloc( &host, SIZE, hipHostMallocMapped );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc - mapped" << std::endl;
            return 1;
        }
        hipHostGetDevicePointer( &mapped_host, host, 0 /*has to be zero "for now" */ );
        
        hipEventRecord( start );
            hipMemcpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost/device to device - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   
            
        hipEventRecord( start );       
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost/device to device - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;       
        
        hipFree( dev );
        hipHostFree( host );
    }
    //
    {
        real_t *dev, *host;
        hipMalloc( &dev, SIZE );
        const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocWriteCombined );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc - write combining" << std::endl;
            return 1;
        }

        hipEventRecord( start );
            hipMemcpy( dev, host, SIZE, hipMemcpyHostToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost - write combining:         " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   
        
        hipEventRecord( start );
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost - write combining:         " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;       
        
        hipFree( dev );
        hipHostFree( host );
    }
    //
    {
        real_t *dev, *host, *mapped_host;
        hipMalloc( &dev, SIZE );
        const hipError_t e = hipHostAlloc( &host, SIZE, hipHostMallocMapped | hipHostMallocWriteCombined );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostAlloc - mapped | write combining" << std::endl;
            return 1;
        }
        hipHostGetDevicePointer( &mapped_host, host, 0 /*has to be zero "for now" */ );
        
        hipEventRecord( start );
            hipMemcpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - cudaAllocHost - mapped, write combining: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   

        hipEventRecord( start );
            hipMemcpy( host, dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - cudaAllocHost - mapped, write combining: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl;   


        hipFree( dev );
        hipHostFree( host );
    }

    #if CUDART_VERSION >= 4000
    //
    {
        real_t *dev, *mapped_host;
        hipMalloc( &dev, SIZE );
        std::vector< real_t > host( NUM_ELEMENTS ); 

        const hipError_t e = hipHostRegister( &host[ 0 ], SIZE, hipHostMallocMapped );
        if( e != hipSuccess ) {
            std::cerr << "Error - hipHostRegister - mapped" << std::endl;
            return 1;
        }
        hipHostGetDevicePointer( &mapped_host, &host[ 0 ], 0 /*has to be zero "for now" */ );
        
        hipEventRecord( start );
            hipMemcpy( dev, mapped_host, SIZE, hipMemcpyDeviceToDevice );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nHost to device - hipHostRegister - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl; 

        hipEventRecord( start );
            hipMemcpy( &host[ 0 ], dev, SIZE, hipMemcpyDeviceToHost );
        hipEventRecord( stop );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &elapsed, start, stop );
        std::cout << "\nDevice to host - hipHostRegister - mapped: " << GBs( NUM_ELEMENTS, elapsed ) << std::endl; 

        hipFree( dev );
        hipHostUnregister( &host[ 0 ] );
    }
    #endif
    //
    {
        void *hostSrc, *hostDest;
        hostSrc  = malloc( NUM_ELEMENTS * sizeof( real_t ) );
                hostDest = malloc( NUM_ELEMENTS * sizeof( real_t ) );
                timespec t1,t2;
                //timeval t1, t2;
                double elapsedTime;
                //gettimeofday( &t1, 0 );
                clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &t1 );
                memcpy( hostDest, hostSrc, NUM_ELEMENTS * sizeof( real_t ) );
                clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &t2 );
                //gettimeofday( &t2, 0 );
                //elapsedTime = ( t2.tv_sec - t1.tv_sec ) * 1000.0 +
                //              ( t2.tv_usec - t1.tv_usec ) / 1000.0;     
                elapsedTime = ( t2.tv_sec - t1.tv_sec ) * 1000.0 +
                              ( t2.tv_nsec - t1.tv_nsec ) / 1000000;

                std::cout << "\nHost to Host: " << 2 * GBs( NUM_ELEMENTS, float( elapsedTime ) ) << std::endl;  

        free( hostSrc  );
        free( hostDest );
    }
    {
        void *hostSrc, *hostDest;
        hostSrc  = aligned_malloc( NUM_ELEMENTS * sizeof( real_t ), 256 );
                hostDest = aligned_malloc( NUM_ELEMENTS * sizeof( real_t ), 256 );
                timespec t1,t2;
                //timeval t1, t2;
                double elapsedTime;
                //gettimeofday( &t1, 0 );
                clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &t1 );
                memcpy( hostDest, hostSrc, NUM_ELEMENTS * sizeof( real_t ) );
                clock_gettime( CLOCK_PROCESS_CPUTIME_ID, &t2 );
                //gettimeofday( &t2, 0 );
                //elapsedTime = ( t2.tv_sec - t1.tv_sec ) * 1000.0 +
                //              ( t2.tv_usec - t1.tv_usec ) / 1000.0;     
                elapsedTime = ( t2.tv_sec - t1.tv_sec ) * 1000.0 +
                              ( t2.tv_nsec - t1.tv_nsec ) / 1000000;

                std::cout << "\nHost to Host - aligned: " << 2 * GBs( NUM_ELEMENTS, float( elapsedTime ) ) << std::endl;  

        aligned_free( hostSrc  );
        aligned_free( hostDest );
    }
    return 0;
}
